#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <assert.h>
#include "DataFormats/DetId/interface/DetId.h"
#include "DataFormats/EcalDetId/interface/EBDetId.h"

__global__ void test_gen_detid(DetId* id) {
    DetId did;
    *id = did;
}

__global__ void test_gen_ecal_detid(EBDetId *id) {
    EBDetId did(10, 80);
    *id = did;

    // trigger functions on the device
    did.iphi();
    did.ieta();
    did.zside();
    did.subdet();
    did.ietaAbs();
    did.ism();
    did.im();
    did.ic();
    did.iphiSM();
    did.ietaSM();
    did.positiveZ();
    did.numberBySM();
    did.approxEta();
}

void test_detid() {
    // test det ids
    DetId h_id, h_id_test;
    DetId h_test0{1};
    DetId *d_id;

    hipMalloc((void**)&d_id, sizeof(DetId));
    hipMemcpy(d_id, &h_id, sizeof(DetId), hipMemcpyHostToDevice);
    test_gen_detid<<<1,1>>>(d_id);
    hipMemcpy(&h_id_test, d_id, sizeof(DetId), hipMemcpyDeviceToHost);
    
    assert(h_id_test == h_id);
    assert(h_id != h_test0);
}

void test_ecal_detid() {
    EBDetId h_id;
    EBDetId h_id_test0{10, 80};
    EBDetId *d_id;

    hipMalloc((void**)&d_id, sizeof(EBDetId));
    hipMemcpy(d_id, &h_id, sizeof(EBDetId), hipMemcpyHostToDevice);
    test_gen_ecal_detid<<<1,1>>>(d_id);
    hipMemcpy(&h_id, d_id, sizeof(EBDetId), hipMemcpyDeviceToHost);

    std::cout << h_id_test0 << std::endl;
    std::cout << h_id << std::endl;
    assert(h_id_test0 == h_id);
}

int main(int argc, char** argv) {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    std::cout << "nDevices = " << nDevices << std::endl;

    // test det id functionality
    if (nDevices>0)
        test_detid();

    // test ecal det ids
    if (nDevices>0)
        test_ecal_detid();

    return 0;
}
